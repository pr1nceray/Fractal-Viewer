#include "hip/hip_runtime.h"
#include "Viewer.cuh"



Viewer::Viewer() {
	Set_Defaults();

	hipError_t err;

	img_host = new sf::Uint8[sz_total];

	err = hipMalloc((void**)&dest_dev, sz_total * sizeof(sf::Uint8));

	if (err != hipSuccess) {
		fprintf(stderr, "hipMalloc launch failed: %s\n", hipGetErrorString(err));
		free_resources(dest_dev, img_host);		//MAYBE CHANGE?
		dest_dev = nullptr;
		img_host = nullptr;
	}

}

Viewer::~Viewer() {
	if (dest_dev != nullptr && img_host != nullptr) {
		free_resources(dest_dev, img_host);
	}
}
void Viewer::display() {
	hipError_t err;
	while (window.isOpen()) {
		if (precise) {
			call_kernel<float>();
		}
		else {
			call_kernel<double>();
		}
		Check_Events(); //adjust scaling
		sync();
		update_display();
	}
}

void Viewer::Set_Defaults() {
	res_x = 1024;
	res_y = 512;
	num_channels = 4;
	sz_total = res_x * res_y * num_channels;

	img_display.create(res_x, res_y);
	sprite = sf::Sprite(img_display);

	window.create(sf::VideoMode(1024, 512), "Fractal Viewer");
	window.setFramerateLimit(60); // maybe remove?
	mode = 0;

	last_mouse = { -1, -1 };
}

void Viewer::Check_Events() {
	sf::Event e;

	while (window.pollEvent(e)) {
		if (e.type == sf::Event::Closed) {
			window.close();
		}
		if (e.type == sf::Event::MouseWheelMoved) {
			float moved = e.mouseWheel.delta;	//scaling needed here!
			if (moved > 0) {
				scale = scale / sqrt(abs(moved) + .025);
			}
			else {
				scale = scale * sqrt(abs(moved) + .025);
				scale = std::min(scale, 4.0);
			}
		}
	}
	Check_Keyboard();
	Check_Mouse();
	if (abs(center_x) > 2) {
		center_x = 2 * (center_x < 0 ? -1 : 1);
	}
	if (abs(center_y) > 1) {
		center_y = 1 * (center_y < 0 ? -1 : 1);
	}

}

void Viewer::Check_Keyboard() {
	int dir_y = (sf::Keyboard::isKeyPressed(sf::Keyboard::Up) ? 1 : 0) + (sf::Keyboard::isKeyPressed(sf::Keyboard::Down) ? -1 : 0);
	int dir_x = (sf::Keyboard::isKeyPressed(sf::Keyboard::Right) ? 1 : 0) + (sf::Keyboard::isKeyPressed(sf::Keyboard::Left) ? -1 : 0);
	center_y += dir_y * (scale / 64.0);
	center_x += dir_x * (scale / 64.0);
	
	if (sf::Keyboard::isKeyPressed(sf::Keyboard::P)) {
		stbi_write_png("screenshot.png", res_x, res_y, num_channels, img_host, num_channels * res_x);
	}
}

void Viewer::Check_Mouse() {
	if (sf::Mouse::isButtonPressed(sf::Mouse::Left)) {
		auto mouse_pos = sf::Mouse::getPosition(window);

		if (last_mouse.x != -1) {
			center_x -= ((mouse_pos.x - last_mouse.x) / 1024.0) * scale * 2; //flipped for x, multiplied by 2 for aspect ratio
			center_y += ((mouse_pos.y - last_mouse.y) / 512.0) * scale;
		}
		last_mouse = mouse_pos;
	}
	else {
		last_mouse.x = -1; last_mouse.y = -1;
	}
}

template<typename T>
void Viewer::call_kernel() {
	//Determine_ends<float> << <entire_block, xyblock >> > (dest_dev, scale, center_x, center_y, max_iters);

	switch (mode)
	{
	case 0: {
		Determine_ends<T> << <entire_block, xyblock >> > (dest_dev, (T)scale, (T)center_x, (T)center_y, max_iters);
		break;
	}
	case 1: {
		//juliaSet
	}
	default:
		break;
	}
}

void Viewer::sync() {
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		free_resources(dest_dev, img_host);
	}
	hipMemcpy(img_host, dest_dev, sz_total * sizeof(sf::Uint8), hipMemcpyDeviceToHost);
}

void Viewer::update_display() {
	img_display.update(img_host);
	sprite.setTexture(img_display);
	window.clear();
	window.draw(sprite);
	window.display();
}

void Viewer::resize() {
	//update info	
}

void Viewer::free_resources(uint8_t* dest_dev, sf::Uint8* tmp) {
	hipFree(dest_dev);
	delete[] tmp;

}
